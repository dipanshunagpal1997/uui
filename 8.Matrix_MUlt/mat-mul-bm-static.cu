
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define Width 4
#define TILE_WIDTH 2
__global__ void mat_mul(int *a, int *b,int *ab, int width)
{
	// shorthand
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x, by = blockIdx.y;
	// allocate tiles in __shared__ memory
	__shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_b[TILE_WIDTH][TILE_WIDTH];
	// calculate the row & col index
	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;
	int result = 0;

	// loop over the tiles of the input in phases
	for(int p = 0; p < width/TILE_WIDTH; ++p)
	{
		// collaboratively load tiles into __shared__
		s_a[ty][tx] = a[row*width + (p*TILE_WIDTH + tx)];
		s_b[ty][tx] = b[(p*TILE_WIDTH + ty)*width + col];
		__syncthreads();
		// dot product between row of s_a and col of s_b
		for(int k = 0; k < TILE_WIDTH; ++k)
		result += s_a[ty][k] * s_b[k][tx];
		__syncthreads();
	}
	ab[row*width+col] = result;
}


int main()
{
    int mat_size=Width*Width*sizeof(int);	//Calculate memory size required for float matrix
    //int tot_elements=Width*Width;
    int M[Width][Width],N[Width][Width],P[Width][Width];	// Host matrix pointers

	int i=0,j=0;
	int *Md,*Nd,*Pd;		//Matrix Pointer on device memoryi.e GPU

	printf("\nEntering elements for matrix");
	for(i=0;i<Width;i++)
	{
		for(j=0;j<Width;j++)
		{
			M[i][j]=1;
			N[i][j]=1;
		}

	}

	printf("Matrix M=\n ");
	for(i=0;i<Width;i++)
	{
		for(j=0;j<Width;j++)
		{
			printf("%d\t",M[i][j]);
		}
		printf("\n");
	}
	printf("Matrix N=\n ");
	for(i=0;i<Width;i++)
	{
		for(j=0;j<Width;j++)
		{
			printf("%d\t",N[i][j]);
		}
		printf("\n");
	}
	hipMalloc((void**)&Md,mat_size);		//Allocate memory on device global memory
	hipMemcpy(Md,M,mat_size,hipMemcpyHostToDevice);	//Copy matrix data from host to device memory
	hipMalloc((void**)&Nd,mat_size);
	hipMemcpy(Nd,N,mat_size,hipMemcpyHostToDevice);
		hipMalloc((void**)&Pd,mat_size);

	dim3 dimGrid(TILE_WIDTH,TILE_WIDTH);	//Variable for threads arrangement in a block.
	dim3 dimBlock(Width/TILE_WIDTH,Width/TILE_WIDTH);		//Variable for blocks arrangement in a grid.

	mat_mul<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);	//Kernel invocation with grid and block specification in angle brackets


	hipMemcpy(P,Pd,mat_size,hipMemcpyDeviceToHost);	//Copy resultant matrix from device to host
	//display the resultant matrix
	printf("Product=\n ");
	for(i=0;i<Width;i++)
	{
		for(j=0;j<Width;j++)
		{
			printf("%d\t",P[i][j]);
		}
		printf("\n");
	}
	//Free device memory
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

}

