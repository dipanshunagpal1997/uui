#include "hip/hip_runtime.h"


 #include <stdio.h>
 #include <stdlib.h>

 #define MAX_THREADS 128
 #define N 10
 #define MAX_LEVELS	300

 int*	host_values;
 int*	device_values;

 __global__ static void quicksort(int* values)
{
	int pivot, L, R;
	int idx =  threadIdx.x + blockIdx.x * blockDim.x;
	int start[MAX_LEVELS];
	int end[MAX_LEVELS];

	start[idx] = idx;
	end[idx] = N - 1;
	while (idx >= 0)
	{
		L = start[idx];
		R = end[idx];
		if (L < R)
		{
			pivot = values[L];
			while (L < R)
			{
				while (values[R] >= pivot && L < R)
					R--;
				if(L < R)
					values[L++] = values[R];
				while (values[L] < pivot && L < R)
					L++;
				if (L < R)
					values[R--] = values[L];
			}
			values[L] = pivot;
			start[idx + 1] = L + 1;
			end[idx + 1] = end[idx];
			end[idx++] = L;
			if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1])
			{

				int tmp = start[idx];
				start[idx] = start[idx - 1];
				start[idx - 1] = tmp;

				tmp = end[idx];
				end[idx] = end[idx - 1];
				end[idx - 1] = tmp;
	        }

		}
		else
		{
			idx--;
		}
	}
}


 int main()
 {

 	size_t size = N * sizeof(int);

 	host_values = (int*)malloc(size);

    hipMalloc((void**)&device_values, size);

    const  int cThreadsPerBlock = 128;

    for (int x = 0; x < N; ++x)
    {
    	printf("Enter Number:");
    	scanf("%d",&host_values[x]);
    }

	hipMemcpy(device_values, host_values, size, hipMemcpyHostToDevice) ;

	quicksort <<< MAX_THREADS / cThreadsPerBlock, MAX_THREADS / cThreadsPerBlock, cThreadsPerBlock >>> (device_values);

	hipMemcpy(host_values, device_values, size, hipMemcpyDeviceToHost) ;

    for (int x = 0; x < N; ++x)
    {
    	printf("\n%d",host_values[x]);
    }

	hipFree(device_values) ;
 	free(host_values);

 	hipDeviceReset();
}

 /*
  *
  *
  * Last login: Tue Apr  7 14:18:46 2015 from 10.80.0.65
echo $PWD'>'
/bin/sh -c "cd \"/tmp/nsight-debug\";\"/tmp/nsight-debug/test\"";exit
cuda-admin@cuda-admin:~$ echo $PWD'>'
/home/cuda-admin>
cuda-admin@cuda-admin:~$ /bin/sh -c "cd \"/tmp/nsight-debug\";\"/tmp/nsight-debu g/test\"";exit
Enter Number:
9
9

Enter Number:
8
8


Enter Number:
7
7

Enter Number:
6
6

Enter Number:
5
5


Enter Number:
4
4


Enter Number:
3
3


Enter Number:
2
2

Enter Number:
1
1

Enter Number:
10
10


1
2
3
4
5
6
7
8
9
10

logout
  *
  */

